#include "hip/hip_runtime.h"
#include <particle_simulator.hpp>
#include "EoS.h"
#include "param.h"
#include "mathfunc.h"
#include "kernel.h"
#include "class.h"
#include "kernel.cuh"
#include "class_device.hpp"

struct kernel_t_device{
	__device__ static real pow8(const real x){
		const real x2 = x  * x;
		const real x4 = x2 * x2;
		return x4 * x4;
	}
	__device__ static real plus(const real x){
		return (x > 0.0) ? x : 0.0;
	}
	__device__ static real pow7(const real x){
		real x2 = x * x;
		real x4 = x2 * x2;
		return x4 * x2 * x;
	}
	//W
	__device__ real W(const real r, const real h) const{
		const real H = supportRadius() * h;
		const real s = r / H;
		real r_value;
		//r_value = (1.0 + s * (8.0 + s * (25.0 + s * (32.0)))) * pow8(plus(1.0 - s));
		r_value = __fmaf_rn(__fmaf_rn(__fmaf_rn(s, 32.0, 25.0), s, 8.0), s, 1.0) * pow8(plus(1.0 - s));
		r_value *= (1365./64.) / (H * H * H * M_PI);
		return r_value;
	}
	//gradW
	__device__ real gradW(const real r, const real h) const{
		const real H = supportRadius() * h;
		const real s = r / H;
		real r_value;
		r_value = pow7(plus(1.0 - s)) * (plus(1.0 - s) * (8.0 + s * (50.0 + s * (96.0))) - 8.0 * (1.0 + s * (8.0 + s * (25.0 + s * (32.0)))));
		r_value *= (1365./64.) / (H * H * H * M_PI);
		return r_value / (H + 0.01 * h);
	}
	__device__ static real supportRadius(){
		return 3.5;
	}
};

static struct{
	int *ni_displc_d, *nj_displc_d, *ni_displc_h, *nj_displc_h;
	Drvt::EpiDev *epi_d, *epi_h;
	Drvt::EpjDev *epj_d, *epj_h;
	Drvt::ForceDev *res_d, *res_h;
}drvt_host;

static struct{
	int *ni_displc_d, *nj_displc_d, *ni_displc_h, *nj_displc_h;
	Hydr::EpiDev *epi_d, *epi_h;
	Hydr::EpjDev *epj_d, *epj_h;
	Hydr::ForceDev *res_d, *res_h;
}hydr_host;

__global__ void deviceCalcDensity(const Dens::EpiDev *epi, const int *ni_displc, const Dens::EpjDev *epj, const int *nj_displc, Dens::ForceDev *dens){
	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	kernel_t_device kernel;
	const Dens::EpiDev& ith = epi[id];
	const int j_head = nj_displc[ith.id_walk];
	const int j_tail = nj_displc[ith.id_walk + 1];
	real dens_buf = 0.0;
	for(int j = j_head ; j < j_tail ; ++ j){
		const Dens::EpjDev& jth = epj[j];
		const double3 dr = make_double3(jth.rx - ith.rx, jth.ry - ith.ry, jth.rz - ith.rz);
		//const real r = sqrtf(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z);
		const real r = __fsqrt_rn(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z);
		dens_buf += jth.mass * kernel.W(r, ith.smth);
	}
	dens[id].dens = max(dens_buf, 5.0);
}

__global__ void deviceCalcDerivative(const Drvt::EpiDev *epi, const int *ni_displc, const Drvt::EpjDev *epj, const int *nj_displc, Drvt::ForceDev *force){
	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	kernel_t_device kernel;
	const Drvt::EpiDev& ith = epi[id];
	const int j_head = nj_displc[ith.id_walk];
	const int j_tail = nj_displc[ith.id_walk + 1];
	double4 force_buf = make_double4(0.0, 0.0, 0.0, 0.0);
	for(int j = j_head ; j < j_tail ; ++ j){
		const Drvt::EpjDev& jth = epj[j];
		double3 dr = make_double3(jth.rx - ith.rx, jth.ry - ith.ry, jth.rz - ith.rz);
		double3 dv = make_double3(jth.vx - ith.vx, jth.vy - ith.vy, jth.vz - ith.vz);
		const real r = sqrtf(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z) + 1.0e-4;
		const real rinv = 1.0 / r;
		const real drdv = dr.x * dv.x + dr.y * dv.y + dr.z * dv.z;

		const real jth_mass_ith_abs_gradW = jth.mass * kernel.gradW(r, ith.smth) * rinv;
		force_buf.w += - drdv * jth_mass_ith_abs_gradW;
		force_buf.x += - (dr.y * dv.z - dr.z * dv.y) * jth_mass_ith_abs_gradW;
		force_buf.y += - (dr.z * dv.x - dr.x * dv.z) * jth_mass_ith_abs_gradW;
		force_buf.z += - (dr.x * dv.y - dr.y * dv.x) * jth_mass_ith_abs_gradW;
	}
	force[id].rot_vx = force_buf.x;
	force[id].rot_vy = force_buf.y;
	force[id].rot_vz = force_buf.z;
	force[id].div_v  = force_buf.w;
}

__global__ void deviceCalcHydroForce(const Hydr::EpiDev *epi, const int *ni_displc, const Hydr::EpjDev *epj, const int *nj_displc, Hydr::ForceDev *force){
	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	kernel_t_device kernel;
	const Hydr::EpiDev& ith = epi[id];
	const int j_head = nj_displc[ith.id_walk];
	const int j_tail = nj_displc[ith.id_walk + 1];

	double v_sig_max = 0.0;
	double4 force_buf = make_double4(0.0, 0.0, 0.0, 0.0);
	const real ith_pres_over_dens2 = ith.pres / (ith.dens * ith.dens);
	for(int j = j_head ; j < j_tail ; ++ j){
		const Hydr::EpjDev& jth = epj[j];
		const double3 dr = make_double3(jth.rx - ith.rx, jth.ry - ith.ry, jth.rz - ith.rz);
		const double3 dv = make_double3(jth.vx - ith.vx, jth.vy - ith.vy, jth.vz - ith.vz);
		/*
		const real r  = sqrtf(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + 1.0e-8f);
		const real rinv = __fdiv_rn(1.0f, r);
		*/
		const real r2 = __fmaf_rn(dr.x, dr.x, __fmaf_rn(dr.y, dr.y, __fmaf_rn(dr.z, dr.z, 1.0e-4)));
		const real rinv = __frsqrt_rn(r2);
		const real r = r2 * rinv;
		const real drdv = dr.x * dv.x + dr.y * dv.y + dr.z * dv.z;
		//AV
		const real w_ij = (drdv < 0.0) ? drdv * rinv : 0.0;
		const real v_sig = ith.snds + jth.snds - 3.0 * w_ij;
		v_sig_max = (v_sig_max < v_sig) ? v_sig : v_sig_max;
		const real AV = - 0.5 * v_sig * w_ij / (0.5 * (ith.dens + jth.dens));
		/*
		const real ith_abs_gradW = kernel.gradW(r, ith.smth);
		const real jth_abs_gradW = kernel.gradW(r, jth.smth);
		const real abs_gradW_rinv = 0.5f * (ith_abs_gradW + jth_abs_gradW) * rinv;
		*/
		const real abs_gradW_rinv = 0.5 * kernel.gradW(r, 0.5 * (ith.smth + jth.smth)) * rinv;
		const double3 gradW = make_double3(abs_gradW_rinv * dr.x, abs_gradW_rinv * dr.y, abs_gradW_rinv * dr.z);
		const real acc = jth.mass * (ith_pres_over_dens2 + jth.pres / (jth.dens * jth.dens) + AV);
		force_buf.x += acc * gradW.x;
		force_buf.y += acc * gradW.y;
		force_buf.z += acc * gradW.z;
		force_buf.w += jth.mass * (ith_pres_over_dens2 + 0.5 * AV) * (dv.x * gradW.x + dv.y * gradW.y + dv.z * gradW.z);
	}
	force[id].ax = force_buf.x;
	force[id].ay = force_buf.y;
	force[id].az = force_buf.z;
	force[id].eng_dot = force_buf.w;
	force[id].dt = PARAM::C_CFL * 2.0 * ith.smth / v_sig_max;
}

namespace DENS{
	static struct{
		Dens::ForceDev *res;
		int *ni_displc, *nj_displc;
		Dens::EpiDev *epi;
		Dens::EpjDev *epj;
	}host, dev;

	int DispatchKernel(const PS::S32 tag, const int n_walk, const STD::EPI::Dens* epi[], const int* n_epi, const STD::EPJ::Dens* epj[], const int* n_epj){
		static bool isFirst = true;
		if(isFirst == true){
			std::cout << "Alloc Cuda Vars.." << std::endl;
			(hipMalloc    ((void**)&dev.ni_displc, (N_WALK_LIMIT + 1) * sizeof(int)));
			(hipMalloc    ((void**)&dev.nj_displc, (N_WALK_LIMIT + 1) * sizeof(int)));
			(hipHostMalloc((void**)&host.ni_displc, (N_WALK_LIMIT + 1) * sizeof(int)));
			(hipHostMalloc((void**)&host.nj_displc, (N_WALK_LIMIT + 1) * sizeof(int)));
			(hipMalloc    ((void**)&dev.epi, NI_LIMIT * sizeof(Dens::EpiDev)));
			(hipMalloc    ((void**)&dev.epj, NJ_LIMIT * sizeof(Dens::EpjDev)));
			(hipMalloc    ((void**)&dev.res, NI_LIMIT * sizeof(Dens::ForceDev)));
			(hipHostMalloc((void**)&host.epi, NI_LIMIT * sizeof(Dens::EpiDev)));
			(hipHostMalloc((void**)&host.epj, NJ_LIMIT * sizeof(Dens::EpjDev)));
			(hipHostMalloc((void**)&host.res, NI_LIMIT * sizeof(Dens::ForceDev)));
			isFirst = false;
		}
		host.ni_displc[0] = host.nj_displc[0] = 0;
		for(std::size_t i = 0; i < n_walk ; ++ i){
			host.ni_displc[i+1] = host.ni_displc[i] + n_epi[i];
			host.nj_displc[i+1] = host.nj_displc[i] + n_epj[i];
		}
		const PS::S32 ni_total = host.ni_displc[n_walk];
		if(ni_total >= NI_LIMIT){
			std::cout << ni_total << " >= " << NI_LIMIT << std::endl;
			assert(ni_total < NI_LIMIT);
		}
		const int ni_total_reg = host.ni_displc[n_walk] + ((ni_total % N_THREAD_GPU != 1) ? (N_THREAD_GPU - (ni_total % N_THREAD_GPU)) : 0);
		//make data for the device on the host
		int cnt = 0;
		int cnt_j = 0;
		for(std::size_t walk = 0 ; walk < n_walk ; ++ walk){
			for(std::size_t i = 0 ; i < n_epi[walk] ; ++ i){
				host.epi[cnt].rx = epi[walk][i].pos.x;
				host.epi[cnt].ry = epi[walk][i].pos.y;
				host.epi[cnt].rz = epi[walk][i].pos.z;
				host.epi[cnt].mass = epi[walk][i].mass;
				host.epi[cnt].smth = epi[walk][i].smth;
				host.epi[cnt].id_walk = walk;
				++ cnt;
			}
			for(std::size_t j = 0 ; j < n_epj[walk] ; ++ j){
				host.epj[cnt_j].rx = epj[walk][j].pos.x;
				host.epj[cnt_j].ry = epj[walk][j].pos.y;
				host.epj[cnt_j].rz = epj[walk][j].pos.z;
				host.epj[cnt_j].mass = epj[walk][j].mass;
				host.epj[cnt_j].smth = epj[walk][j].smth;
				++ cnt_j;
			}
		}

		(hipMemcpy(dev.epi, host.epi, ni_total_reg * sizeof(Dens::EpiDev), hipMemcpyHostToDevice));
		(hipMemcpy(dev.epj, host.epj, cnt_j * sizeof(Dens::EpjDev), hipMemcpyHostToDevice));
		(hipMemcpy(dev.ni_displc, host.ni_displc, (n_walk + 1) * sizeof(int), hipMemcpyHostToDevice));
		(hipMemcpy(dev.nj_displc, host.nj_displc, (n_walk + 1) * sizeof(int), hipMemcpyHostToDevice));

		const int n_grid = ni_total_reg / N_THREAD_GPU + ((ni_total_reg % N_THREAD_GPU == 0) ? 0 : 1);
		dim3 size_grid(n_grid, 1, 1);
		dim3 size_thread(N_THREAD_GPU, 1, 1);
		deviceCalcDensity<<<size_grid, size_thread>>> (dev.epi, dev.ni_displc, dev.epj, dev.nj_displc, dev.res);
		return 0;
	}

	int RetrieveKernel(const PS::S32 tag, const PS::S32 n_walk, const PS::S32* ni, STD::RESULT::Dens* force[]){
		int ni_tot = 0;
		for(int i = 0 ; i < n_walk ; ++ i){
			ni_tot += ni[i];
		}
		(hipMemcpy(host.res, dev.res, ni_tot * sizeof(Dens::ForceDev), hipMemcpyDeviceToHost));
		int cnt = 0;
		for(int walk = 0 ; walk < n_walk ; ++ walk){
			for(int i = 0 ; i < ni[walk] ; ++ i){
				force[walk][i].dens = max(host.res[cnt].dens, 5.0);
				++ cnt;
			}
		}
		return 0;
	}
};

int DrvtDispatchKernel(const PS::S32 tag, const int n_walk, const STD::EPI::Drvt* epi[], const int* n_epi, const STD::EPJ::Drvt* epj[], const int* n_epj){
	static bool isFirst = true;
	if(isFirst == true){
		std::cout << "Alloc Cuda Vars.." << std::endl;
		(hipMalloc    ((void**)&drvt_host.ni_displc_d, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipMalloc    ((void**)&drvt_host.nj_displc_d, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipHostMalloc((void**)&drvt_host.ni_displc_h, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipHostMalloc((void**)&drvt_host.nj_displc_h, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipMalloc    ((void**)&drvt_host.epi_d, NI_LIMIT * sizeof(Drvt::EpiDev)));
		(hipMalloc    ((void**)&drvt_host.epj_d, NJ_LIMIT * sizeof(Drvt::EpjDev)));
		(hipMalloc    ((void**)&drvt_host.res_d, NI_LIMIT * sizeof(Drvt::ForceDev)));
		(hipHostMalloc((void**)&drvt_host.epi_h, NI_LIMIT * sizeof(Drvt::EpiDev)));
		(hipHostMalloc((void**)&drvt_host.epj_h, NJ_LIMIT * sizeof(Drvt::EpjDev)));
		(hipHostMalloc((void**)&drvt_host.res_h, NI_LIMIT * sizeof(Drvt::ForceDev)));
		isFirst = false;
	}
	drvt_host.ni_displc_h[0] = drvt_host.nj_displc_h[0] = 0;
	for(std::size_t i = 0; i < n_walk ; ++ i){
		drvt_host.ni_displc_h[i+1] = drvt_host.ni_displc_h[i] + n_epi[i];
		drvt_host.nj_displc_h[i+1] = drvt_host.nj_displc_h[i] + n_epj[i];
	}
	const PS::S32 ni_total = drvt_host.ni_displc_h[n_walk];
	if(ni_total >= NI_LIMIT){
		std::cout << ni_total << " >= " << NI_LIMIT << std::endl;
		assert(ni_total < NI_LIMIT);
	}
	if(drvt_host.nj_displc_h[n_walk] >= NJ_LIMIT){
		std::cout << drvt_host.nj_displc_h[n_walk] << " >= " << NJ_LIMIT << std::endl;
		assert(drvt_host.nj_displc_h[n_walk] < NJ_LIMIT);
	}

	const int ni_total_reg = drvt_host.ni_displc_h[n_walk] + ((ni_total % N_THREAD_GPU != 1) ? (N_THREAD_GPU - (ni_total % N_THREAD_GPU)) : 0);
	//make data for device on host
	int cnt = 0;
	int cnt_j = 0;
	for(std::size_t walk = 0 ; walk < n_walk ; ++ walk){
		for(std::size_t i = 0 ; i < n_epi[walk] ; ++ i){
			drvt_host.epi_h[cnt].rx = epi[walk][i].pos.x;
			drvt_host.epi_h[cnt].ry = epi[walk][i].pos.y;
			drvt_host.epi_h[cnt].rz = epi[walk][i].pos.z;
			drvt_host.epi_h[cnt].vx = epi[walk][i].vel.x;
			drvt_host.epi_h[cnt].vy = epi[walk][i].vel.y;
			drvt_host.epi_h[cnt].vz = epi[walk][i].vel.z;
			drvt_host.epi_h[cnt].dens = epi[walk][i].dens;
			drvt_host.epi_h[cnt].smth = epi[walk][i].smth;
			drvt_host.epi_h[cnt].id_walk = walk;
			++ cnt;
		}
		for(std::size_t j = 0 ; j < n_epj[walk] ; ++ j){
			drvt_host.epj_h[cnt_j].rx = epj[walk][j].pos.x;
			drvt_host.epj_h[cnt_j].ry = epj[walk][j].pos.y;
			drvt_host.epj_h[cnt_j].rz = epj[walk][j].pos.z;
			drvt_host.epj_h[cnt_j].vx = epj[walk][j].vel.x;
			drvt_host.epj_h[cnt_j].vy = epj[walk][j].vel.y;
			drvt_host.epj_h[cnt_j].vz = epj[walk][j].vel.z;
			drvt_host.epj_h[cnt_j].mass = epj[walk][j].mass;
			drvt_host.epj_h[cnt_j].smth = epj[walk][j].smth;
			++ cnt_j;
		}
	}

	(hipMemcpy(drvt_host.epi_d, drvt_host.epi_h, ni_total_reg * sizeof(Drvt::EpiDev), hipMemcpyHostToDevice));
	(hipMemcpy(drvt_host.epj_d, drvt_host.epj_h, cnt_j * sizeof(Drvt::EpjDev), hipMemcpyHostToDevice));
	(hipMemcpy(drvt_host.ni_displc_d, drvt_host.ni_displc_h, (n_walk + 1) * sizeof(int), hipMemcpyHostToDevice));
	(hipMemcpy(drvt_host.nj_displc_d, drvt_host.nj_displc_h, (n_walk + 1) * sizeof(int), hipMemcpyHostToDevice));

	const int n_grid = ni_total_reg / N_THREAD_GPU + ((ni_total_reg % N_THREAD_GPU == 0) ? 0 : 1);
	dim3 size_grid(n_grid, 1, 1);
	dim3 size_thread(N_THREAD_GPU, 1, 1);
	deviceCalcDerivative<<<size_grid, size_thread>>> (drvt_host.epi_d, drvt_host.ni_displc_d, drvt_host.epj_d, drvt_host.nj_displc_d, drvt_host.res_d);
	return 0;
}

int DrvtRetrieveKernel(const PS::S32 tag, const PS::S32 n_walk, const PS::S32* ni, STD::RESULT::Drvt* force[]){
	int ni_tot = 0;
	for(int i = 0 ; i < n_walk ; ++ i){
		ni_tot += ni[i];
	}
	(hipMemcpy(drvt_host.res_h, drvt_host.res_d, ni_tot * sizeof(Drvt::ForceDev), hipMemcpyDeviceToHost));
	int cnt = 0;
	for(int walk = 0 ; walk < n_walk ; ++ walk){
		for(int i = 0 ; i < ni[walk] ; ++ i){
			force[walk][i].div_v   = drvt_host.res_h[cnt].div_v;
			force[walk][i].rot_v.x = drvt_host.res_h[cnt].rot_vx;
			force[walk][i].rot_v.y = drvt_host.res_h[cnt].rot_vy;
			force[walk][i].rot_v.z = drvt_host.res_h[cnt].rot_vz;
			++ cnt;
		}
	}
	return 0;
}

int HydrDispatchKernel(const PS::S32 tag, const int n_walk, const STD::EPI::Hydro** epi, const int* n_epi, const STD::EPJ::Hydro** epj, const int* n_epj){
	static bool isFirst = true;
	if(isFirst == true){
		std::cout << "Alloc Cuda Vars.." << std::endl;
		(hipMalloc    ((void**)&hydr_host.ni_displc_d, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipMalloc    ((void**)&hydr_host.nj_displc_d, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipHostMalloc((void**)&hydr_host.ni_displc_h, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipHostMalloc((void**)&hydr_host.nj_displc_h, (N_WALK_LIMIT + 1) * sizeof(int)));
		(hipMalloc    ((void**)&hydr_host.epi_d, NI_LIMIT * sizeof(Hydr::EpiDev)));
		(hipMalloc    ((void**)&hydr_host.epj_d, NJ_LIMIT * sizeof(Hydr::EpjDev)));
		(hipMalloc    ((void**)&hydr_host.res_d, NI_LIMIT * sizeof(Hydr::ForceDev)));
		(hipHostMalloc((void**)&hydr_host.epi_h, NI_LIMIT * sizeof(Hydr::EpiDev)));
		(hipHostMalloc((void**)&hydr_host.epj_h, NJ_LIMIT * sizeof(Hydr::EpjDev)));
		(hipHostMalloc((void**)&hydr_host.res_h, NI_LIMIT * sizeof(Hydr::ForceDev)));
		isFirst = false;
	}
	hydr_host.ni_displc_h[0] = hydr_host.nj_displc_h[0] = 0;
	for(std::size_t i = 0; i < n_walk ; ++ i){
		hydr_host.ni_displc_h[i+1] = hydr_host.ni_displc_h[i] + n_epi[i];
		hydr_host.nj_displc_h[i+1] = hydr_host.nj_displc_h[i] + n_epj[i];
	}
	const PS::S32 ni_total = hydr_host.ni_displc_h[n_walk];
	const int ni_total_reg = hydr_host.ni_displc_h[n_walk] + ((ni_total % N_THREAD_GPU != 0) ? (N_THREAD_GPU - (ni_total % N_THREAD_GPU)) : 0);
	//make data for device on host
	int cnt = 0;
	int cnt_j = 0;
	for(std::size_t walk = 0 ; walk < n_walk ; ++ walk){
		for(std::size_t i = 0 ; i < n_epi[walk] ; ++ i){
			hydr_host.epi_h[cnt].rx      = epi[walk][i].pos.x;
			hydr_host.epi_h[cnt].ry      = epi[walk][i].pos.y;
			hydr_host.epi_h[cnt].rz      = epi[walk][i].pos.z;
			hydr_host.epi_h[cnt].vx      = epi[walk][i].vel.x;
			hydr_host.epi_h[cnt].vy      = epi[walk][i].vel.y;
			hydr_host.epi_h[cnt].vz      = epi[walk][i].vel.z;
			hydr_host.epi_h[cnt].dens    = epi[walk][i].dens;
			hydr_host.epi_h[cnt].pres    = epi[walk][i].pres;
			hydr_host.epi_h[cnt].snds    = epi[walk][i].snds;
			hydr_host.epi_h[cnt].smth    = epi[walk][i].smth;
			hydr_host.epi_h[cnt].Bal     = epi[walk][i].Bal;
			hydr_host.epi_h[cnt].id_walk = walk;
			hydr_host.epi_h[cnt].grad_smth = epi[walk][i].grad_smth;
			++ cnt;
		}
		for(std::size_t j = 0 ; j < n_epj[walk] ; ++ j){
			hydr_host.epj_h[cnt_j].rx   = epj[walk][j].pos.x;
			hydr_host.epj_h[cnt_j].ry   = epj[walk][j].pos.y;
			hydr_host.epj_h[cnt_j].rz   = epj[walk][j].pos.z;
			hydr_host.epj_h[cnt_j].vx   = epj[walk][j].vel.x;
			hydr_host.epj_h[cnt_j].vy   = epj[walk][j].vel.y;
			hydr_host.epj_h[cnt_j].vz   = epj[walk][j].vel.z;
			hydr_host.epj_h[cnt_j].dens = epj[walk][j].dens;
			hydr_host.epj_h[cnt_j].pres = epj[walk][j].pres;
			hydr_host.epj_h[cnt_j].snds = epj[walk][j].snds;
			hydr_host.epj_h[cnt_j].mass = epj[walk][j].mass;
			hydr_host.epj_h[cnt_j].smth = epj[walk][j].smth;
			hydr_host.epj_h[cnt_j].Bal  = epj[walk][j].Bal;
			hydr_host.epj_h[cnt_j].grad_smth = epj[walk][j].grad_smth;
			++ cnt_j;
		}
	}

	(hipMemcpy(hydr_host.epi_d, hydr_host.epi_h, ni_total_reg * sizeof(Hydr::EpiDev), hipMemcpyHostToDevice));
	(hipMemcpy(hydr_host.epj_d, hydr_host.epj_h, cnt_j * sizeof(Hydr::EpjDev), hipMemcpyHostToDevice));
	(hipMemcpy(hydr_host.ni_displc_d, hydr_host.ni_displc_h, (n_walk + 1) * sizeof(int), hipMemcpyHostToDevice));
	(hipMemcpy(hydr_host.nj_displc_d, hydr_host.nj_displc_h, (n_walk + 1) * sizeof(int), hipMemcpyHostToDevice));

	const int n_grid = ni_total_reg / N_THREAD_GPU + ((ni_total_reg % N_THREAD_GPU == 0) ? 0 : 1);
	dim3 size_grid(n_grid, 1, 1);
	dim3 size_thread(N_THREAD_GPU, 1, 1);
	deviceCalcHydroForce<<<size_grid, size_thread>>> (hydr_host.epi_d, hydr_host.ni_displc_d, hydr_host.epj_d, hydr_host.nj_displc_d, hydr_host.res_d);
	return 0;
}

int HydrRetrieveKernel(const PS::S32 tag, const PS::S32 n_walk, const PS::S32* ni, STD::RESULT::Hydro** force){
	int ni_tot = 0;
	for(int i = 0 ; i < n_walk ; ++ i){
		ni_tot += ni[i];
	}
	(hipMemcpy(hydr_host.res_h, hydr_host.res_d, ni_tot * sizeof(Hydr::ForceDev), hipMemcpyDeviceToHost));
	int cnt = 0;
	for(int walk = 0 ; walk < n_walk ; ++ walk){
		for(int i = 0 ; i < ni[walk] ; ++ i){
			force[walk][i].acc.x = hydr_host.res_h[cnt].ax;
			force[walk][i].acc.y = hydr_host.res_h[cnt].ay;
			force[walk][i].acc.z = hydr_host.res_h[cnt].az;
			force[walk][i].eng_dot = hydr_host.res_h[cnt].eng_dot;
			force[walk][i].dt = hydr_host.res_h[cnt].dt;
			++ cnt;
		}
	}
	return 0;
}


